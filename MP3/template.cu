#include "hip/hip_runtime.h"

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH (4)
// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

  int tx = threadIdx.x; int ty = threadIdx.y;
  int Col = blockIdx.x * TILE_WIDTH + tx;
  int Row = blockIdx.y * TILE_WIDTH + ty;
  
  float ans = 0;
  
    for (int i = 0; i < ceil(numAColumns * 1.0 / TILE_WIDTH); ++i){
    if (Row < numARows && (i * TILE_WIDTH + tx) < numAColumns)
    {
       subTileA[ty][tx] = A[numAColumns * Row + i * TILE_WIDTH + tx];
    }
    else subTileA[ty][tx] = 0;
   
    if (Col < numBColumns && (i * TILE_WIDTH + ty) < numBRows)
    {
      subTileB[ty][tx] = B[numBColumns * (TILE_WIDTH * i + ty) + Col];  
    }
    else subTileB[ty][tx] = 0;
    
    // all threads has to load its corresponding memory
    __syncthreads();
    
    // boundary check 
    for (int j = 0; j < TILE_WIDTH; j++)
    {
      ans += subTileA[ty][j] * subTileB[j][tx];
    }
    __syncthreads();
  }
  // C element updated inside the for loop of tile index
  if (Row < numCRows && Col < numCColumns)
    {
      C[Row * numCColumns + Col]  = ans;  
    }
  return ;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(sizeof(float) * numCRows * numCColumns);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceA, sizeof(float) * numARows * numAColumns);
  hipMalloc((void **) &deviceB, sizeof(float) * numBRows * numBColumns);
  hipMalloc((void **) &deviceC, sizeof(float) * numCRows * numCColumns);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // int TILE_WIDTH = 16;
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 DimGrid(ceil((numCColumns * 1.0)/TILE_WIDTH), ceil((numCRows * 1.0)/TILE_WIDTH) ,1);
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows,numAColumns, numBRows,numBColumns, numCRows,numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
